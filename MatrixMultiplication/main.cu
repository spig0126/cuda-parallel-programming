#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  main.cu
 *
 *    Description:  Matrix Multiplication
 *
 *        Version:  1.0
 *        Created:  01/18/2022
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Yeonhee Jung
 *
 * =====================================================================================
 */

#include <assert.h>
#include "mkCuda.h"
#include "mkClockMeasure.h"

const int A_H = 200;
const int A_W = 3000;
const int B_H = A_W;
const int B_W = 200;
const unsigned int MAX_NUM = 10;
const int MAX_ITER = 10;

unsigned int matrixA[A_H * A_W];
unsigned int matrixB[B_H * B_W];
unsigned int cpuOut[A_H * B_W];
unsigned int gpuOut[A_H * B_W];

void generateRandomValues(unsigned int *input, const int rowSize, const int colSize){
	for(int i = 0; i < rowSize; i++){
		for(int j = 0; j < colSize; j++){
			input[i * colSize + j] = (unsigned int) float(rand())/float(RAND_MAX) * MAX_NUM;
		}
	}
}

void printMatrixValue(const unsigned int *input, const int rowSize, const int colSize){
	printf("Print Matrix \n -----------\n");
	for(int i = 0; i < rowSize; i++){
		for(int j = 0; j < colSize; j++){
			printf("%u\t", input[i * colSize + j]);
		}
		printf("\n");
	}
	printf("--------\n");
}

bool compareMatrix(const unsigned int *inputA, const unsigned int *inputB, const int rowSize, const int colSize){
	bool ret = true;
	for(int i = 0; i < rowSize * colSize; i++){
		if(inputA[i] != inputB[i]){
			ret = false;
			break;
		}
	}
	return ret;
}

void cpuMatrixMul(const unsigned int *h_a, const unsigned int *h_b, unsigned int *h_c, const int aRowSize, const int aColSize, const int bRowSize, const int bColSize){
	assert(aColSize == bRowSize);

	for (int r = 0; r < aRowSize; r++){
		for (int c = 0; c < bColSize; c++){
			int temp = 0;
			for (int i = 0; i < aColSize; i++)
			{
				temp += h_a[r*aColSize + i] * h_b[i*bColSize + c];
			}
			h_c[r * bColSize + c] = temp;
		}
	}
}

__global__
void gpuMatrixMul(unsigned int *d_a, unsigned int *d_b, unsigned int *d_c, const int aRowSize, const int aColSize, const int bRowSize, const int bColSize){
	assert(aColSize == bRowSize);

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if(row<aRowSize && col<bColSize){
		int index = row * bColSize + col;
		d_c[index] = 0;
		for (int i = 0; i < aColSize; i++){
			d_c[index] += d_a[row * aColSize + i] * d_b[i * bColSize + col];
			//intf("(%d/%d)=(%d/%d) %d\n", row, col, d_c[index], (row*aColSize+i), (i*bRowSize + col));
		}
	}
}

int main(){
	srand((unsigned int)time(NULL));
	generateRandomValues(matrixA, A_H, A_W);
	generateRandomValues(matrixB, B_H, B_W);

	//MK: GPU Memory 
	unsigned int *d_a, *d_b, *d_c;
	size_t matrixSizeA = sizeof(unsigned int) * A_H * A_W;
	size_t matrixSizeB = sizeof(unsigned int) * B_H * B_W;
	size_t matrixSizeC = sizeof(unsigned int) * A_H * B_W;
	
	//allocate memory in device
	hipError_t err = hipMalloc((void **) &d_a, matrixSizeA);
	checkCudaError(err);
	err = hipMalloc((void **) &d_b, matrixSizeB);
	checkCudaError(err);
	err = hipMalloc((void **) &d_c, matrixSizeC);
	checkCudaError(err);

	err = hipMemcpy(d_a, matrixA, matrixSizeA, hipMemcpyHostToDevice);
	checkCudaError(err);
	err = hipMemcpy(d_b, matrixB, matrixSizeB, hipMemcpyHostToDevice);
	checkCudaError(err);

	//MK: Thread Num
	//const int tbSize = 256;
	const int threads = 16;
	dim3 gridSize(ceil((float)(A_H) / (float)threads), ceil((float)(B_W) / (float)threads), 1);
	dim3 blockSize(threads, threads, 1);

	//MK: Time Measurement
	mkClockMeasure *ckCpu = new mkClockMeasure("CPU CODE");
	ckCpu->clockReset();
	
	mkClockMeasure *ckGpu = new mkClockMeasure("GPU CODE");
	ckGpu->clockReset();

	for(int i = 0; i < MAX_ITER; i++){
		ckCpu->clockResume();
		cpuMatrixMul(matrixA, matrixB, cpuOut, A_H, A_W, B_H, B_W);
		ckCpu->clockPause();

		ckGpu->clockResume();
		gpuMatrixMul<<<gridSize, blockSize>> >(d_a, d_b, d_c, A_H, A_W, B_H, B_W);
		err=hipDeviceSynchronize();
		ckGpu->clockPause();
		checkCudaError(err);

	}

	err = hipMemcpy(gpuOut, d_c, matrixSizeC, hipMemcpyDeviceToHost);
	checkCudaError(err);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	if(compareMatrix(cpuOut, gpuOut, A_H, B_W)){
		ckCpu->clockPrint();
		ckGpu->clockPrint();
	}else{
		printf("ERROR: Two Matrices are not same\n");
	}

	/*printMatrixValue(matrixA, A_H, A_W);
	printMatrixValue(matrixB, B_H, B_W);
	printMatrixValue(cpuOut, A_H, B_W);
	printMatrixValue(gpuOut, A_H, B_W);*/
}

