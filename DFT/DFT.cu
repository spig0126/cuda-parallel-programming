#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  DFT.cu
 *
 *    Description:  DFT
 *
 *        Version:  1.0
 *        Created:  07/20/2022 
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Yeonhee Jung
 *   Organization:  EWHA Womans Unversity
 *
 * =====================================================================================
 */

#include <stdio.h>
#include <math.h>
#include <complex.h>
#include "comp.cuh"
#include "mkClockMeasure.h"
#include "mkCuda.h"


const int SAMPLING_RATE = 131072;
const int N = SAMPLING_RATE;
const int FREQ_NUM = 3;
const int MAX_ITER = 1;
int db_bytesize = N * sizeof(double);
int comp_bytesize = N * 2 * sizeof(double);

double freq_amp_ph[FREQ_NUM][3] = {{1, 3, 0}, {4, 1, 0}, {7, 0.5, 0}};
double sample_points[N], freq[N], sig[N];
//cpu
Comp x[N];
double idft_sig[N], amp[N];
//gpu
Comp gpu_x[N];
double gpu_idft_sig[N], gpu_amp[N];

// thread, grid size */ 
int thread = 64;
int tbSize = N/thread;
dim3 gridSize(tbSize, 1, 1);
dim3 blockSize(thread, 1, 1);

void create_sample_points(double* sample_points){
    for (int i = 0; i<N; i++){
        sample_points[i] = (double) i/SAMPLING_RATE;
        freq[i] = i;

        // printf("%lf\n", sample_points[i]);
    }
}

void generate_sig(double* sample_points, double* sig){
    for (int s_i = 0; s_i < N; s_i++){
        sig[s_i] = 0;
        for (int f_i = 0; f_i < FREQ_NUM; f_i++)
        {
            sig[s_i] += freq_amp_ph[f_i][1] * sin(2 * M_PI * freq_amp_ph[f_i][0] * sample_points[s_i]);
        }
    }
}

bool compareResult(double* a, double* b, int size){
    double epsilon = 0.000001;

    for(int i=0; i<size; i++){
        if(fabs(a[i] - b[i]) > epsilon){
            // printf("a; %lf, \t b: %lf\n", a[i], b[i]);
            return false;
        }
    }
    return true;
}
void printData(double* x, double* y, int size, const char* x_label, const char* y_label){
    for(int i=0; i<size; i++){
        printf("%s: %lf\t%s: %lf\n", x_label, x[i], y_label, y[i]);
    }
}

void save_data(double* x_values, double* y_values, const char* path){
    FILE *dataf = fopen(path, "w");
    for (int i=0; i<N; i++){
        fprintf(dataf, "%lf %lf\n", x_values[i], y_values[i]);
    }
}

void cpu_dft(mkClockMeasure* ck){
    ck->clockResume();

    double exp = 2 * M_PI / N;

    for (int k = 0; k<N; k++){
        x[k].i = 0;
        x[k].r = 0;
        for (int n = 0; n < N; n++)
        {
            double bn = exp * k * n;
            x[k].i -= sig[n] * sin(bn);
            x[k].r += sig[n] * cos(bn);
        }
        amp[k] = 2 * sqrt(pow(x[k].r, 2) + pow(x[k].i, 2)) / N;
        // printf("frequency: %f\tamplitude: %f\n", freq[k], x[k]);
        // printf("[CPU] k: %d\txi: %lf\t, xr: %lf\n", k, xi[k], xr[k]);

    }
    ck->clockPause();
}

__global__ void gpu_dft(Comp* d_x, double* d_amp, double* d_sig, int N){
    //1 freq = 1 thread
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    double xi=0, xr=0;
    double exp = 2 * M_PI * k / N;


    for(int n=0; n<N; n++){
        double bn = exp * n;
        xi -= d_sig[n] * sin(bn);
        xr += d_sig[n] * cos(bn);
    }
    d_x[k].i = xi;
    d_x[k].r = xr;
    d_amp[k] =  2 * sqrt(pow(xr, 2) + pow(xi, 2)) / N;

    // printf("frequency: %d\tamplitude: %lf\t sig: %f\n", k, d_x[k], d_sig[k]);
    // printf("k: %d\n", k);
}

void cal_gpu_dft(mkClockMeasure* ck_mem_transfer, mkClockMeasure* ck_kernel, mkClockMeasure* ck_exec){
    // allocate device memory 
    Comp *d_x;
    double *d_amp, *d_sig;
    hipError_t err = hipHostMalloc((void**)&d_x, comp_bytesize);
    checkCudaError(err);
    err = hipHostMalloc((void**)&d_amp, db_bytesize);
    checkCudaError(err);
    err = hipHostMalloc((void**)&d_sig, db_bytesize);

    ck_exec->clockResume();

    // memory transfer: host -> device
    ck_mem_transfer->clockResume();
    err = hipMemcpy(d_sig, sig, db_bytesize, hipMemcpyHostToDevice);
    checkCudaError(err);
    ck_mem_transfer->clockPause();


    // launch kernel
    ck_kernel->clockResume();
    gpu_dft<<<gridSize, blockSize>>>(d_x, d_amp, d_sig, N); 
    ck_kernel->clockPause();

    // memory transfer: device -> host 
    ck_mem_transfer->clockResume();
    err = hipMemcpy(gpu_amp, d_amp, db_bytesize, hipMemcpyDeviceToHost);
    checkCudaError(err);
    err = hipMemcpy(gpu_x, d_x, comp_bytesize, hipMemcpyDeviceToHost);
    ck_mem_transfer->clockPause();

    ck_exec->clockPause();

    // free device memory
    hipFree(d_amp);
    hipFree(d_sig);
}

void cpu_idft(mkClockMeasure *ck){
    ck->clockResume();
    for (int n = 0; n < N; n++)
    {
        idft_sig[n] = 0;
        for (int k = 0; k < N; k++)
        {
            double bn = 2 * M_PI * k * n / N;
            idft_sig[n] += x[k].r * cos(bn) + x[k].i * sin(bn);
        }
        idft_sig[n] /= N;
        // printf("t: %d\tamplitude: %lf\n", n, idft_sig[n]);
    }
    ck->clockPause();

}

__global__ void gpu_idft(double* d_idft_sig, Comp* d_x, int N){
    // 1 signal at sample point n = 1 thread
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    double exp = 2 * M_PI * n / N;
    double sig = 0;
    

    for(int k=0; k<N; k++){
        double bn = exp * k;
        sig += d_x[k].r * cos(bn) + d_x[k].i * sin(bn);
    }

    d_idft_sig[n] = sig / N;   
    // printf("t: %d\tamplitude: %lf\n", n, d_idft_sig[n]);
    // printf("[GPU] k: %d\txi: %lf\t, xr: %lf\n", n, d_xi[n], d_xr[n]);
}

void cal_gpu_idft(mkClockMeasure* ck_mem_transfer, mkClockMeasure* ck_kernel, mkClockMeasure* ck_exec){
    // allocate device memory 
    Comp *d_x;
    double *d_idft_sig;
    hipError_t err = hipHostMalloc((void**)&d_idft_sig, db_bytesize);
    err = hipHostMalloc((void**)&d_x, comp_bytesize);
    checkCudaError(err);

    ck_exec->clockResume();

    // memory transfer: host -> device
    ck_mem_transfer->clockResume();
    err = hipMemcpy(d_x, gpu_x, comp_bytesize, hipMemcpyHostToDevice);
    ck_mem_transfer->clockPause();

    // launch kernel
    ck_kernel->clockResume();
    gpu_idft<<<gridSize, blockSize>>>(d_idft_sig, d_x, N); 
    ck_kernel->clockPause();

    // memory transfer: device -> host 
    ck_mem_transfer->clockResume();
    err = hipMemcpy(gpu_idft_sig, d_idft_sig, db_bytesize, hipMemcpyDeviceToHost);
    checkCudaError(err);
    ck_mem_transfer->clockPause();

    ck_exec->clockPause();

    // ree device memory 
    hipFree(d_idft_sig);
}



int main(void){
    // int sampling_rates = [64, 256, 1024, 4096]
    printf("SAMPLING_RATE : %d\nMAX_ITERATION : %d\n\n", SAMPLING_RATE, MAX_ITER);

    /* set basic data */
    create_sample_points(sample_points);
    generate_sig(sample_points, sig);

    /* create clocks */
    mkClockMeasure *ckCpu_dft = new mkClockMeasure("CPU - DFT");
    mkClockMeasure *ckCpu_idft = new mkClockMeasure("CPU - IDFT");
    mkClockMeasure *ckGpu_dft_mem_transfer = new mkClockMeasure("MEMORY TRANSFER");
    mkClockMeasure *ckGpu_dft_kernels = new mkClockMeasure("KERNELS");
    mkClockMeasure *ckGpu_dft = new mkClockMeasure("GPU - DFT TOTAL");
    mkClockMeasure *ckGpu_idft_mem_transfer = new mkClockMeasure("MEMORY TRANSFER");
    mkClockMeasure *ckGpu_idft_kernels = new mkClockMeasure("KERNELS");
    mkClockMeasure *ckGpu_idft = new mkClockMeasure("GPU - IDFT TOTAL");
    ckCpu_dft->clockReset(), ckCpu_idft->clockReset();
    ckGpu_dft_mem_transfer->clockReset(), ckGpu_dft_kernels->clockReset(), ckGpu_dft->clockReset();
    ckGpu_idft_mem_transfer->clockReset(), ckGpu_idft_kernels->clockReset(), ckGpu_idft->clockReset();

    for(int i=0; i<MAX_ITER; i++){
        // /* CPU - DFT */
        // cpu_dft(ckCpu_dft);

        // /* CPU - IDFT */
        // cpu_idft(ckCpu_idft);

        /* GPU - DFT */
        cal_gpu_dft(ckGpu_dft_mem_transfer, ckGpu_dft_kernels, ckGpu_dft);

        /* GPU - IDFT */  
        cal_gpu_idft(ckGpu_idft_mem_transfer, ckGpu_idft_kernels, ckGpu_idft);
    }




    /* print DFT performance */
    // if(compareResult(amp, gpu_amp, N)){
    if(true){
        printf("-------------------[CPU] DFT ---------------------\n");
		ckCpu_dft->clockPrint();
        printf("\n-------------------[GPU] DFT ---------------------\n");
        ckGpu_dft_mem_transfer->clockPrint();
        ckGpu_dft_kernels->clockPrint();
		ckGpu_dft->clockPrint();
    }
    else{
        printf("ERROR: DFT results are not the same\n\n");
    }

    /* print Inverse DFT performance */
    // if(compareResult(idft_sig, gpu_idft_sig, N)){
    if(true){
        printf("\n\n-------------------[CPU] Inverse DFT ---------------------\n");
        ckCpu_idft->clockPrint();
        printf("\n-------------------[GPU] Inverse DFT ---------------------\n");
        ckGpu_idft_mem_transfer->clockPrint();
        ckGpu_idft_kernels->clockPrint();
		ckGpu_idft->clockPrint();
    }
    else{
        printf("ERROR: Inverse DFT results are not the same\n\n");
    }

    /* save data */
    save_data(sample_points, sig,  "data/original_signal.txt");
    save_data(freq, amp, "data/cpu_dft_frequencies.txt");
    save_data(freq, gpu_amp, "data/gpu_dft_frequencies.txt");
    save_data(sample_points, idft_sig, "data/idft_signal.txt");
    save_data(sample_points, gpu_idft_sig, "data/gpu_idft_signal.txt");
}