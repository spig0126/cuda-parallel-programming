#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  DFT.cu
 *
 *    Description:  DFT
 *
 *        Version:  1.0
 *        Created:  07/20/2022 
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Yeonhee Jung
 *   Organization:  EWHA Womans Unversity
 *
 * =====================================================================================
 */

#include <stdio.h>
#include <math.h>
#include <complex.h>
#include "comp.h"
#include "mkClockMeasure.h"
#include "mkCuda.h"


const int SAMPLING_RATE = 4096;
const int N = SAMPLING_RATE;
const int FREQ_NUM = 3;
const int MAX_ITER = 100;


double freq_amp_ph[FREQ_NUM][3] = {{1, 3, 0}, {4, 1, 0}, {7, 0.5, 0}};
double sample_points[N], freq[N], sig[N];
//cpu
Comp x[N];
double idft_sig[N], amp[N];
//gpu
Comp gpu_x[N];
double gpu_idft_sig[N], gpu_amp[N];

__constant__ double d_sig[N];

void create_sample_points(double* sample_points){
    for (int i = 0; i<N; i++){
        sample_points[i] = (double) i/SAMPLING_RATE;
        freq[i] = i;

        // printf("%lf\n", sample_points[i]);
    }
}

void generate_sig(double* sample_points, double* sig){
    for (int s_i = 0; s_i < N; s_i++){
        sig[s_i] = 0;
        for (int f_i = 0; f_i < FREQ_NUM; f_i++)
        {
            sig[s_i] += freq_amp_ph[f_i][1] * sin(2 * M_PI * freq_amp_ph[f_i][0] * sample_points[s_i]);
        }
    }
}

bool compareResult(double* a, double* b, int size){
    double epsilon = 0.000001f;

    for(int i=0; i<size; i++){
        if(fabs(a[i] - b[i]) < epsilon){
            // printf("a; %lf, \t b: %lf\n", a[i], b[i]);
            return true;
        }
    }
    return false;
}

void printData(double* x, double* y, int size, const char* x_label, const char* y_label){
    for(int i=0; i<size; i++){
        printf("%s: %lf\t%s: %lf\n", x_label, x[i], y_label, y[i]);
    }
}

void save_data(double* x_values, double* y_values, const char* path){
    FILE *dataf = fopen(path, "w");
    for (int i=0; i<N; i++){
        fprintf(dataf, "%lf %lf\n", x_values[i], y_values[i]);
    }
}

void cpu_dft(mkClockMeasure* ck){
    ck->clockResume();

    double exp = 2 * M_PI / N;

    for (int k = 0; k<N; k++){
        x[k].i = 0;
        x[k].r = 0;
        for (int n = 0; n < N; n++)
        {
            double bn = exp * k * n;
            x[k].i -= sig[n] * sin(bn);
            x[k].r += sig[n] * cos(bn);
        }
        amp[k] = 2 * sqrt(pow(x[k].r, 2) + pow(x[k].i, 2)) / N;
        // printf("frequency: %f\tamplitude: %f\n", freq[k], x[k]);
        // printf("[CPU] k: %d\txi: %lf\t, xr: %lf\n", k, xi[k], xr[k]);

    }
    ck->clockPause();
}

__global__ void gpu_dft(Comp* d_x, double* d_amp, int N){
    //1 freq = 1 thread
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    double xi=0, xr=0;
    double exp = 2 * M_PI * k / N;


    for(int n=0; n<N; n++){
        double bn = exp * n;
        xi -= d_sig[n] * sin(bn);
        xr += d_sig[n] * cos(bn);
    }
    d_x[k].i = xi;
    d_x[k].r = xr;
    d_amp[k] =  2 * sqrt(pow(xr, 2) + pow(xi, 2)) / N;
    // printf("frequency: %d\tamplitude: %lf\t sig: %f\n", k, d_x[k], d_sig[k]);
    // printf("k: %d\n", k);

}

void cpu_idft(mkClockMeasure *ck){
    ck->clockResume();
    for (int n = 0; n < N; n++)
    {
        idft_sig[n] = 0;
        for (int k = 0; k < N; k++)
        {
            double bn = 2 * M_PI * k * n / N;
            idft_sig[n] += x[k].r * cos(bn) + x[k].i * sin(bn);
        }
        idft_sig[n] /= N;
        // printf("t: %d\tamplitude: %lf\n", n, idft_sig[n]);
    }
    ck->clockPause();

}

__global__ void gpu_idft(double* d_idft_sig, Comp* d_x, int N){
    // 1 signal at sample point n = 1 thread
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    double exp = 2 * M_PI * n / N;
    double sig = 0;
    

    for(int k=0; k<N; k++){
        double bn = exp * k;
        sig += d_x[k].r * cos(bn) + d_x[k].i * sin(bn);
    }

    d_idft_sig[n] = sig / N;

    // printf("t: %d\tamplitude: %lf\n", n, d_idft_sig[n]);
    // printf("[GPU] k: %d\txi: %lf\t, xr: %lf\n", n, d_xi[n], d_xr[n]);
}


int main(void){
    // int sampling_rates = [64, 256, 1024, 4096]
    printf("SAMPLING_RATE : %d\nMAX_ITERATION : %d\n\n", SAMPLING_RATE, MAX_ITER);

    /* set basic data */
    create_sample_points(sample_points);
    generate_sig(sample_points, sig);

    /* create clocks */
    mkClockMeasure *ckCpu_dft = new mkClockMeasure("CPU - DFT"),  *ckCpu_idft = new mkClockMeasure("CPU - IDFT"), *ckGpu_dft = new mkClockMeasure("GPU - DFT"), *ckGpu_idft = new mkClockMeasure("GPU - IDFT");
    ckCpu_dft->clockReset(), ckCpu_idft->clockReset(), ckGpu_dft->clockReset(), ckGpu_idft->clockReset();

    /* allocate device memory */ 
    Comp *d_x;
    double *d_idft_sig, *d_amp;
    int db_bytesize = N * sizeof(double);
    int comp_bytesize = N * 2 * sizeof(double);

    hipError_t err = hipMalloc((void**)&d_idft_sig, db_bytesize);
    checkCudaError(err);
    err = hipMalloc((void**)&d_x, comp_bytesize);
    checkCudaError(err);
    err = hipMalloc((void**)&d_amp, db_bytesize);
    checkCudaError(err);

    /* set thread, grid size */ 
    int thread = 256;
    int tbSize = N/thread;
    dim3 gridSize(tbSize, 1, 1);
    dim3 blockSize(thread, 1, 1);


    for(int i=0; i<MAX_ITER; i++){
        /* CPU - DFT */
        cpu_dft(ckCpu_dft);

        /* GPU - DFT */
        ckGpu_dft->clockResume();

        // memory transfer: host -> device
        err = hipMemcpyToSymbol(HIP_SYMBOL(d_sig), sig, db_bytesize); //constant memory
        checkCudaError(err);

        gpu_dft<<<gridSize, blockSize>>>(d_x, d_amp, N); 

        // memory transfer: device -> host 
        err = hipMemcpy(gpu_x, d_x, comp_bytesize, hipMemcpyDeviceToHost);
        checkCudaError(err);
        err = hipMemcpy(gpu_amp, d_amp, db_bytesize, hipMemcpyDeviceToHost);
        checkCudaError(err);

        ckGpu_dft->clockPause();


        /* CPU - IDFT */
        cpu_idft(ckCpu_idft);

        /* GPU - IDFT */
        ckGpu_idft->clockResume();

        gpu_idft<<<gridSize, blockSize>>>(d_idft_sig, d_x, N); 

        // memory transfer: device -> host 
        err = hipMemcpy(gpu_idft_sig, d_idft_sig, db_bytesize, hipMemcpyDeviceToHost);
	    checkCudaError(err);

        ckGpu_idft->clockPause();
    }


    /* free device memory */
    hipFree(d_idft_sig);
    hipFree(d_x);
    hipFree(d_amp);

    /* print DFT performance */
    if(compareResult(amp, gpu_amp, N)){
        printf("-------------------[CPU] DFT ---------------------\n");
		ckCpu_dft->clockPrint();
        printf("-------------------[GPU] DFT ---------------------\n");
		ckGpu_dft->clockPrint();
    }
    else{
        printf("ERROR: DFT results are not the same\n\n");
    }

    /* print Inverse DFT performance */
    if(compareResult(idft_sig, gpu_idft_sig, N)){
        printf("\n-------------------[CPU] Inverse DFT ---------------------\n");
        ckCpu_idft->clockPrint();
        printf("-------------------[GPU] Inverse DFT ---------------------\n");
		ckGpu_idft->clockPrint();
    }
    else{
        printf("ERROR: Inverse DFT results are not the same\n\n");
    }

    /* save data */
    save_data(sample_points, sig,  "data/original_signal.txt");
    save_data(freq, amp, "data/cpu_dft_frequencies.txt");
    save_data(freq, gpu_amp, "data/gpu_dft_frequencies.txt");
    save_data(sample_points, idft_sig, "data/idft_signal.txt");
    save_data(sample_points, gpu_idft_sig, "data/gpu_idft_signal.txt");
}